#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""

#include <iostream>
#include <iomanip>
#include <opencv/cv.hpp>
#include <opencv2/photo/cuda.hpp>
#include "cudaExamp.h"
#include "misc.h"
#include "Vec3.h"
#include <time.h>

#include "Ray.h"
#include "Sphere.h"
#include "Hittable.h"
#include "HittableList.h"
#include "Material.h"
#include "Camera.h"
#include "WorldGen.h"

#include "RTRendererCUDAQT.h"
#include <QtWidgets/QApplication>

constexpr auto ITER = 50;
constexpr auto SPP = 4;

__global__ void cuHelloWorld()
{
	printf("Hello world");
}

extern "C" void launchKernal()
{
	cuHelloWorld <<<1, 1 >>> ();
}

__device__ Vec3 color(const Ray& r, Hittable** world, int depth, hiprandState* localRandState)
{
	HitRecord rec;
	if ((*world)->hit(r, 0.000001, FLT_MAX, rec, localRandState)) {
		Ray scattered;
		Vec3 attenuation;
		Vec3 emitted = rec.matPtr->emitted(rec.u, rec.v, rec.point);
		if (depth < ITER && rec.matPtr->scatter(r, rec, attenuation, scattered, localRandState)) {
			return emitted + attenuation * color(scattered, world, depth + 1, localRandState);
		}
		else {
			return emitted;
		}
	}
	else {
#ifdef DARKSCENE
		Vec3 c(0, 0, 0);
#else
		Vec3 unit_direction = unitVector(cur_ray.direction);
		double t = 0.5f * (unit_direction.e[1] + 1.0f);
		Vec3 c = (1.0f - t) * Vec3(1.0, 1.0, 1.0) + t * Vec3(0.5, 0.7, 1.0);
#endif
		return c;
	}
}

__device__ Vec3 color(const Ray& r, Hittable** world,hiprandState* localRandState)
{
	Ray cur_ray = r;
	Vec3 cur_attenuation = Vec3(1.0, 1.0, 1.0);
	for (int i = 0; i < ITER; i++) {
		HitRecord rec;
		if ((*world)->hit(cur_ray, 0.001, FLT_MAX, rec, localRandState)) {
			Ray scattered;
			Vec3 attenuation;
			Vec3 emitted = rec.matPtr->emitted(rec.u, rec.v, rec.point);
			if (rec.matPtr->scatter(cur_ray, rec, attenuation, scattered, localRandState)) {
				cur_attenuation *= attenuation;
				cur_attenuation += emitted;
				cur_ray = scattered;
			}
			else {
				return cur_attenuation * emitted;
			}
		}
		else {
#ifdef DARKSCENE
			Vec3 c(0, 0, 0);
#else
			Vec3 unit_direction = unitVector(cur_ray.direction);
			double t = 0.5f * (unit_direction.e[1] + 1.0f);
			Vec3 c = (1.0f - t) * Vec3(1.0, 1.0, 1.0) + t * Vec3(0.5, 0.7, 1.0);
#endif
			return cur_attenuation * c;
		}
	}
	return Vec3(0.05, 0.05, 0.1); // exceeded recursion
}

// Main rander func.
__global__ void render(int frameCount, double* fBuffer, Camera** cam, Hittable** world, hiprandState* randState)  //{b, g, r}, stupid opencv
{
	int i = threadIdx.x + blockIdx.x * blockDim.x;
	int j = threadIdx.y + blockIdx.y * blockDim.y;

#ifdef ALLOWOUTOFBOUND
	if ((i >= MAX_X) || (j >= MAX_Y)) return;
#endif // OUTOFBOUNDDETECT

	int index = j * MAX_X + i;
	hiprandState localRandState = randState[index];
	Vec3 pixel(0, 0, 0);
	if (frameCount)
	{
		pixel.readFrameBuffer(i, j, fBuffer);
		pixel = pixel * pixel;
		pixel *= frameCount;
		pixel *= SPP;
	}
	for (int s = 0; s < SPP; s++) {
		double u = double(i + hiprand_uniform(&localRandState)) / double(MAX_X);
		double v = double(j + hiprand_uniform(&localRandState)) / double(MAX_Y);
		Ray r = (*cam)->getRay(u, v, &localRandState);
		pixel += color(r, world, &localRandState);
	}
	randState[index] = localRandState;
	pixel /= double(SPP);
	pixel /= frameCount + 1.0;
	pixel.e[0] = sqrt(pixel.e[0]);
	pixel.e[1] = sqrt(pixel.e[1]);
	pixel.e[2] = sqrt(pixel.e[2]);

	pixel.writeFrameBuffer(i, j, fBuffer);
}

__global__ void rander_init(hiprandState* randState)
{
	int i = threadIdx.x + blockIdx.x * blockDim.x;
	int j = threadIdx.y + blockIdx.y * blockDim.y;
	if ((i >= MAX_X) || (j >= MAX_Y)) return;
	int pixel_index = j * MAX_X + i;
	hiprand_init(2019+pixel_index, 0, 0, &randState[pixel_index]);
}

int main(int argc, char* argv[])
{
	clock_t clk;
	clk = clock();
	double renderTime;

	std::cout << "Rendering a " << MAX_X << "x" << MAX_Y << " image ";
	std::cout << "in " << BLK_X << "x" << BLK_Y << " blocks, SPP = " <<SPP<<" & depth = "<<ITER<<"\n";

	size_t* pValue = new size_t;
	checkCudaErrors(hipDeviceGetLimit(pValue, hipLimitStackSize));
	std::cout << "Stack size limit: \t\t\t" << *pValue << "Byte. Resizing to 65536...";

	checkCudaErrors(hipDeviceSetLimit(hipLimitStackSize, 1 << 16));
	checkCudaErrors(hipDeviceGetLimit(pValue, hipLimitStackSize));
	std::cout << "...Done. \nStack size limit: \t\t\t" << *pValue << "Byte.\n";

	checkCudaErrors(hipDeviceGetLimit(pValue, hipLimitPrintfFifoSize));
	std::cout << "printf() fifo limit: \t\t\t" << *pValue << "Byte.\n";
	checkCudaErrors(hipDeviceGetLimit(pValue, hipLimitMallocHeapSize));
	std::cout << "Malloc heap size limit: \t\t" << *pValue << "Byte.\n";
	checkCudaErrors(hipDeviceGetLimit(pValue, cudaLimitDevRuntimeSyncDepth));
	std::cout << "cudaLimitDevRuntimeSyncDepth: \t\t" << *pValue << ".\n";
	checkCudaErrors(hipDeviceGetLimit(pValue, cudaLimitDevRuntimePendingLaunchCount));
	std::cout << "cudaLimitDevRuntimePendingLaunchCount: \t" << *pValue << ".\n";
	checkCudaErrors(hipDeviceGetLimit(pValue, cudaLimitMaxL2FetchGranularity));
	std::cout << "cudaLimitMaxL2FetchGranularity: \t" << *pValue << "Byte.\n";

#ifdef _DEBUG
	std::cout << "Warning: Compiled in debug mode and it hurt performance.\n";
#endif

	cv::Mat M(MAX_Y, MAX_X, CV_64FC3, cv::Scalar(0, 0, 0));

	size_t frameBufferSize = 3 * MAX_X * MAX_Y * sizeof(double);
	double* frameBuffer;
	checkCudaErrors(hipMallocManaged((void**)&frameBuffer, frameBufferSize));

	Hittable** cudaList;
	int num_Hittables = 500;
	checkCudaErrors(hipMalloc((void**)&cudaList, num_Hittables * sizeof(Hittable*)));
	Hittable** cudaWorld;
	checkCudaErrors(hipMalloc((void**)&cudaWorld, sizeof(Hittable*)));
	Camera** cudaCam;
	checkCudaErrors(hipMalloc((void**)&cudaCam, sizeof(Camera*)));
	
	double ms = double(clock() - clk);
	std::cout << "Alloc \t\t@ t+ " << ms << " ms.\r\n";

	hiprandState* worldGenRandState;
	checkCudaErrors(hipMalloc((void**)&worldGenRandState, sizeof(hiprandState)));

	cv::Mat em = cv::imread("earthmap.jpg");
	unsigned char* t;
	checkCudaErrors(hipMalloc((void**)&t, sizeof(unsigned char) * em.rows * em.cols * 3));
	checkCudaErrors(hipMemcpy(t, em.data, sizeof(unsigned char) * em.rows * em.cols * 3, hipMemcpyHostToDevice));

	// createRandScene <<<1, 1 >>> (cudaList, cudaWorld, cudaCam, t, em.cols, em.rows, worldGenRandState);
	// createWorld1 <<<1, 1 >>> (cudaList, cudaWorld, cudaCam, worldGenRandState);
	// createCheckerTest <<<1, 1 >>> (cudaList, cudaWorld, cudaCam, worldGenRandState);
	// createCornellBox << <1, 1 >> > (cudaList, cudaWorld, cudaCam, worldGenRandState);
	createCornellSmoke << <1, 1 >> > (cudaList, cudaWorld, cudaCam, worldGenRandState);

	checkCudaErrors(hipGetLastError());
	checkCudaErrors(hipDeviceSynchronize());

	ms = double(clock() - clk);
	std::cout << "WorldGen \t@ t+ " << ms << " ms.\r\n";

	dim3 blocks(MAX_X / BLK_X + 1, MAX_Y / BLK_Y + 1);
	dim3 threads(BLK_X, BLK_Y);

	hiprandState* renderRandomStates;
	checkCudaErrors(hipMalloc((void**)&renderRandomStates, MAX_X * MAX_Y * sizeof(hiprandState)));
	rander_init <<<blocks, threads >>> (renderRandomStates);
	checkCudaErrors(hipGetLastError());
	checkCudaErrors(hipDeviceSynchronize());

	ms = double(clock() - clk);
	double renderStart = ms;
	std::cout << "init rander \t@ t+ " << ms << " ms.\r\n";

	int frameCount = 0;
	while (1)
	{
		renderTime = ms;

		render <<<blocks, threads >>> (frameCount++, frameBuffer, cudaCam, cudaWorld, renderRandomStates);

		checkCudaErrors(hipGetLastError());
		checkCudaErrors(hipDeviceSynchronize());

		ms = double(clock() - clk);
		renderTime = ms - renderTime;
		std::cout << std::fixed << std::setprecision(2) << "Render Time: " << renderTime / 1000.0 << " / " << (ms - renderStart) / 1000.0 / frameCount << " / " << (ms - renderStart)/1000.0 << " s, current SPP = " << frameCount * SPP << "\r\n";

		M.data = (uchar*)frameBuffer;
		cv::imshow("wow", M);
		if (cv::waitKey(1) == 27) break;
	}


	ms = double(clock() - clk);
	std::cout << "Exec time:\t" << ms << " ms.\r\nRender Time:\t" << renderTime << "ms\r\nExpected FPS:\t" << 1000.00 / renderTime;


	hipDeviceReset();

    return 0;
}

